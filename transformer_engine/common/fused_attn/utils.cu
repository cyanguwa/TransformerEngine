#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "transformer_engine/fused_attn.h"
#include "../common.h"
#include "utils.h"

// get QKV layout in enums
NVTE_QKV_Layout get_nvte_qkv_layout(const std::string qkv_layout) {
  if (qkv_layout == "not_interleaved") {
      return NVTE_QKV_Layout::NOT_INTERLEAVED;
  } else if (qkv_layout == "qkv_interleaved") {
      return NVTE_QKV_Layout::QKV_INTERLEAVED;
  } else if (qkv_layout == "kv_interleaved") {
      return NVTE_QKV_Layout::KV_INTERLEAVED;
  } else {
      NVTE_ERROR("Invalid QKV layout. \n");
  }
}

// get bias type in enums
NVTE_Bias_Type get_nvte_bias_type(const std::string bias_type) {
  if (bias_type == "no_bias") {
      return NVTE_Bias_Type::NO_BIAS;
  } else if (bias_type == "pre_scale_bias") {
      return NVTE_Bias_Type::PRE_SCALE_BIAS;
  } else if (bias_type == "post_scale_bias") {
      return NVTE_Bias_Type::POST_SCALE_BIAS;
  } else {
      NVTE_ERROR("Invalid bias type. \n");
  }
}

// get attn mask type in enums
NVTE_Mask_Type get_nvte_mask_type(const std::string mask_type) {
  if (mask_type == "padding") {
      return NVTE_Mask_Type::PADDING;
  } else if (mask_type == "causal") {
      return NVTE_Mask_Type::CAUSAL;
  } else if (mask_type == "no_mask") {
      return NVTE_Mask_Type::NO_MASK;
  } else {
      NVTE_ERROR("Invalid attention mask type. \n");
  }
}

// create NVTETensorPack
void nvte_tensor_pack_create(NVTETensorPack* pack) {
  for (int i = 0; i < pack->MAX_SIZE; i++) {
     pack->tensors[i] = reinterpret_cast<NVTETensor>(new transformer_engine::Tensor);
  }
}

// destroy NVTETensorPack
void nvte_tensor_pack_destroy(NVTETensorPack* pack) {
  for (int i = 0; i < pack->MAX_SIZE; i++) {
     auto *t = reinterpret_cast<transformer_engine::Tensor*>(pack->tensors[i]);
     delete t;
  }
}

// get cuDNN data type
hipdnnDataType_t get_cudnn_dtype(const transformer_engine::DType t) {
  using namespace transformer_engine;
  switch (t) {
    case DType::kFloat16:
      return HIPDNN_DATA_HALF;
    case DType::kFloat32:
      return HIPDNN_DATA_FLOAT;
    case DType::kBFloat16:
      return CUDNN_DATA_BFLOAT16;
    case DType::kFloat8E4M3:
      return CUDNN_DATA_FP8_E4M3;
    case DType::kFloat8E5M2:
      return CUDNN_DATA_FP8_E5M2;
    default:
      NVTE_ERROR("Invalid cuDNN data type. \n");
  }
}

// convert cu_seqlens_q to qkv/o_ragged_offset and actual_seqlens_q
__global__ void cu_seqlens_to_offsets(size_t b, size_t h, size_t d,
                int32_t *cu_seqlens_q, int32_t *actual_seqlens_q,
                int32_t *qkv_ragged_offset, int32_t *o_ragged_offset) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < b) {
    actual_seqlens_q[tid] = cu_seqlens_q[tid + 1] - cu_seqlens_q[tid];
  }
  if (tid < b + 1) {
    qkv_ragged_offset[tid] = cu_seqlens_q[tid] * 3 * h * d;
    o_ragged_offset[tid] = cu_seqlens_q[tid] * h * d;
  }
}


namespace transformer_engine {
namespace fused_attn {

using namespace transformer_engine;

// get matrix strides based on matrix type
void generateMHAStrides(
            int64_t b, int64_t h,
            int64_t s_q, int64_t s_kv,
            int64_t d, int64_t* strideA,
            NVTE_QKV_Layout layout, MHA_Matrix matrix) {
    constexpr int batch_dim_idx   = 0;
    constexpr int head_dim_idx    = 1;
    constexpr int seqlen_dim_idx  = 2;
    constexpr int hidden_dim_idx  = 3;

    constexpr int seqlen_transpose_dim_idx = 3;
    constexpr int hidden_transpose_dim_idx = 2;

    constexpr int seqlen_q_dim_idx = 2;
    constexpr int seqlen_kv_dim_idx = 3;

    switch (matrix) {
        case MHA_Matrix::Q_Matrix:
            if (layout == NVTE_QKV_Layout::QKV_INTERLEAVED) {
                strideA[hidden_dim_idx] = 1;
                strideA[seqlen_dim_idx] = 3 * h * d;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_q * 3 * h * d;
            } else {
                strideA[hidden_dim_idx] = 1;
                strideA[seqlen_dim_idx] = h * d;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_q * h * d;
            }
            break;
        case MHA_Matrix::K_Matrix:
            if (layout == NVTE_QKV_Layout::QKV_INTERLEAVED) {
                strideA[seqlen_dim_idx] = 3 * h * d;
                strideA[hidden_dim_idx] = 1;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * 3 * h * d;
            } else if (layout == NVTE_QKV_Layout::KV_INTERLEAVED) {
                strideA[seqlen_transpose_dim_idx] = 2 * h * d;
                strideA[hidden_transpose_dim_idx] = 1;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * 2 * h * d;
            } else {
                strideA[seqlen_transpose_dim_idx] = h * d;
                strideA[hidden_transpose_dim_idx] = 1;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * h * d;
            }
            break;
        case MHA_Matrix::K_Matrix_Transpose:
            if (layout == NVTE_QKV_Layout::QKV_INTERLEAVED) {
                strideA[seqlen_transpose_dim_idx] = 3 * h * d;
                strideA[hidden_transpose_dim_idx] = 1;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * 3 * h * d;
            } else if (layout == NVTE_QKV_Layout::KV_INTERLEAVED) {
                strideA[seqlen_transpose_dim_idx] = 2 * h * d;
                strideA[hidden_transpose_dim_idx] = 1;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * 2 * h * d;
            } else {
                strideA[seqlen_transpose_dim_idx] = h * d;
                strideA[hidden_transpose_dim_idx] = 1;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * h * d;
            }
            break;
        case MHA_Matrix::V_Matrix:
            if (layout == NVTE_QKV_Layout::QKV_INTERLEAVED) {
                strideA[hidden_dim_idx] = 1;
                strideA[seqlen_dim_idx] = 3 * h * d;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * 3 * h * d;
            } else if (layout == NVTE_QKV_Layout::KV_INTERLEAVED) {
                strideA[hidden_dim_idx] = 1;
                strideA[seqlen_dim_idx] = 2* h * d;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * 2 * h * d;
            } else {
                strideA[hidden_dim_idx] = 1;
                strideA[seqlen_dim_idx] = h * d;
                strideA[head_dim_idx] = d;
                strideA[batch_dim_idx] = s_kv * h * d;
            }
            break;
        case MHA_Matrix::V_Matrix_Transpose:
            if (layout == NVTE_QKV_Layout::QKV_INTERLEAVED) {
                    strideA[hidden_transpose_dim_idx] = 1;
                    strideA[seqlen_transpose_dim_idx] = 3 * h * d;
                    strideA[head_dim_idx] = d;
                    strideA[batch_dim_idx] = s_kv * 3 * h * d;
                } else if (layout == NVTE_QKV_Layout::KV_INTERLEAVED) {
                    strideA[hidden_transpose_dim_idx] = 1;
                    strideA[seqlen_transpose_dim_idx] = 2* h * d;
                    strideA[head_dim_idx] = d;
                    strideA[batch_dim_idx] = s_kv * 2 * h * d;
                } else {
                    strideA[hidden_transpose_dim_idx] = 1;
                    strideA[seqlen_transpose_dim_idx] = h * d;
                    strideA[head_dim_idx] = d;
                    strideA[batch_dim_idx] = s_kv * h * d;
                }
            break;
        case MHA_Matrix::S_Matrix:
            strideA[seqlen_kv_dim_idx] = 1;
            strideA[seqlen_q_dim_idx] = s_kv;
            strideA[head_dim_idx] = s_q * s_kv;
            strideA[batch_dim_idx] = h * s_q * s_kv;
            break;
        case MHA_Matrix::O_Matrix:
            strideA[seqlen_kv_dim_idx] = 1;
            strideA[seqlen_q_dim_idx] = h * d;
            strideA[head_dim_idx] = d;
            strideA[batch_dim_idx] = s_q * h * d;
            break;
    }
}
}  // namespace fused_attn
}  // namespace transformer_engine
